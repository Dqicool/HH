#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_math_constants.h>
#include "GPUScaleB.cuh"
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>
#define Z_MASS 91.1876

#define M_BB_SC     62.5
#define M_TT_SC     62.5
#define MET_SC_1    0.008
#define MET_SC_2    0.0005
#define MET_SC_3    0.000125
#define CHI_SC      0.16

__device__ float getPx(float pt, float phi){
    return pt * cosf(phi);
}

__device__ float getPy(float pt, float phi){
    return pt * sinf(phi);
}

__device__ float getPz(float pt, float eta){
    return pt * sinhf(eta);
}

__device__ float getPScaler(float pt, float eta){
    return pt * coshf(eta);
}

__device__ float getE(float pt, float eta, float m){
    float p = getPScaler(pt, eta);
    return sqrtf(m * m + p * p);
}

__device__ float getMass( float energy, float px, float py, float pz){

    return sqrtf(energy * energy - px * px - py * py - pz * pz);
}

__device__ float getdphi(float v1phi,  float v2phi)
{
    float ret;
    if(fabsf(v1phi - v2phi) > HIP_PI_F)
        ret = ((float)(v1phi > 0) -  (float)(v1phi <= 0)) * (2 * HIP_PI_F - fabsf(v1phi - v2phi));
    else
        ret = v2phi - v1phi;
    return ret;
}

__device__ float getPhiFromPxPy(float px, float py)
{
    return  (float)(px>0) * atanf(py / px) + 
            (float)(px<=0 && py > 0) * (atanf(py / px) + HIP_PI_F) + 
            (float)(px<=0 && py <= 0) * (atanf(py / px) - HIP_PI_F);
}

__device__ float getPsiMbb(float m_bb, float m_baseline, float scale_const_bb)
{
    return (m_bb - m_baseline) * (m_bb - m_baseline) / scale_const_bb;
}

__device__ float getPsiMtt(float m_tt, float m_baseline, float scale_const_tt)
{
    return (m_tt - m_baseline) * (m_tt - m_baseline) / scale_const_tt;
}

__device__ float getPsiMET(float vl_pt, float vh_pt, float tau_pt, float lep_pt, float omega,float sc_1, float sc_2, float sc_3)
{
    float f_tau_l = vl_pt / (vl_pt + lep_pt);
    float f_tau_h = vh_pt / (vh_pt + tau_pt);
    float A_1 = 0.5 * ((f_tau_l - f_tau_h) / (f_tau_l + f_tau_h) + 1);
    float ret = ((float)(A_1 > 0 && A_1 <= 0.8) * (A_1 - 0.8) * (A_1 - 0.8) / sc_1)     +
                ((float)(A_1 > 0.8 && A_1 <= 1) * 0)                                    +
                ((float)(omega > 1)               * (omega - 1) * (omega - 1) / sc_2)   +
                ((float)(omega <= 0)              * (80 + (omega * omega / sc_3)))      ;
    return ret;
}

__device__ float getPsiChi(float chi0, float chi1, float sc_chi)
{
    // calculate on log plane
    float x = log2f(chi0);
    float y = log2f(chi1);

    float x_0      = 9.62213e-02;
    float y_0      = 1.18202e-01;

    float sigma_x  = 4.81883e-01;
    float sigma_y  = 6.37269e-01;

    float theta    = 5.40952e-01;

    // // calculate on linear plane
    // float x = chi0;
    // float y = chi1;

    // float x_0      = 1.07186e+00;
    // float y_0      = 1.05827e+00;

    // float sigma_x  = 3.27060e-01;
    // float sigma_y  = 4.99482e-01;

    // float theta    = 3.50592e-01;

    float ell1 = (powf((x - x_0) * cosf(theta) + (y - y_0) * sinf(theta), 2)) / powf(sigma_x, 2);
    float ell2 = (powf((x - x_0) * sinf(theta) - (y - y_0) * cosf(theta), 2)) / powf(sigma_y, 2);
    float r2 = ell1+ell2;

    // float x         = log2f(chi0);
    // float y         = log2f(chi1);
    // float x_0       = 9.62213e-02;
    // float y_0       = 1.18202e-01;
    // float r2 = powf((x - x_0), 2) + powf((y - y_0), 2);

    return r2 / sc_chi;
}

__global__ void mykernel(   float bjet0_pt, float bjet0_eta,    float bjet0_phi,    float bjet0_m,
                            float bjet1_pt, float bjet1_eta,    float bjet1_phi,    float bjet1_m,
                            float lep0_pt,  float lep0_eta,     float lep0_phi,     float lep0_m,
                            float tau0_pt,  float tau0_eta,     float tau0_phi,     float tau0_m,
                            float met_pt,   float met_eta,      float met_phi,      float met_m,
                            bool * pass, float* score, int N)
{
    int i = (threadIdx.x + blockIdx.x * blockDim.x);
    int j = (threadIdx.y + blockIdx.y * blockDim.y);
    float chi0 = 0.5 + 0.01 * i;
    float chi1 = 0.5 + 0.01 * j;
    // initialise
    size_t sizeof_array = N*N;
    size_t ind = i*N+j;
    pass[ind] = 0;
    score[sizeof_array*1 + ind]  = -999;
    score[sizeof_array*2 + ind]  = -999;
    score[sizeof_array*3 + ind]  = -999;
    score[sizeof_array*4 + ind]  = -999;
    score[ind]   = 100000;
    //mbb calculation
    float bjet0_scaled_pt = bjet0_pt * chi0;
    float bjet1_scaled_pt = bjet1_pt * chi1;

    float met_scaled_px = getPx(met_pt, met_phi) - (getPx(bjet0_scaled_pt, bjet0_phi) - getPx(bjet0_pt, bjet0_phi)) - (getPx(bjet1_scaled_pt, bjet1_phi) - getPx(bjet1_pt, bjet1_phi));
    float met_scaled_py = getPy(met_pt, met_phi) - (getPy(bjet0_scaled_pt, bjet0_phi) - getPy(bjet0_pt, bjet0_phi)) - (getPy(bjet1_scaled_pt, bjet1_phi) - getPy(bjet1_pt, bjet1_phi));
    
    float bjet0_scaled_E = getE(bjet0_scaled_pt, bjet0_eta, bjet0_m);
    float bjet0_scaled_px = getPx(bjet0_scaled_pt, bjet0_phi);
    float bjet0_scaled_py = getPy(bjet0_scaled_pt, bjet0_phi);
    float bjet0_scaled_pz = getPz(bjet0_scaled_pt, bjet0_eta);

    float bjet1_scaled_E =  getE(bjet1_scaled_pt, bjet1_eta, bjet1_m);
    float bjet1_scaled_px = getPx(bjet1_scaled_pt, bjet1_phi);
    float bjet1_scaled_py = getPy(bjet1_scaled_pt, bjet1_phi);
    float bjet1_scaled_pz = getPz(bjet1_scaled_pt, bjet1_eta);


    float m_bb_scaled = getMass(bjet0_scaled_E + bjet1_scaled_E, 
                                bjet0_scaled_px + bjet1_scaled_px,
                                bjet0_scaled_py + bjet1_scaled_py,
                                bjet0_scaled_pz + bjet1_scaled_pz);   
    
    //omega and other
    float dphi_hl = getdphi(tau0_phi, lep0_phi);
    float met_scaled_phi = getPhiFromPxPy(met_scaled_px, met_scaled_py);
    float met_scaled_pt  = sqrtf(met_scaled_py * met_scaled_py + met_scaled_px * met_scaled_px);
    float dphi_hv_scaled = getdphi(tau0_phi, met_scaled_phi);
    float dphi_lv_scaled = getdphi(lep0_phi, met_scaled_phi);
    bool inside_hl_scaled = (dphi_hl * dphi_hv_scaled > 0) && (fabsf(dphi_hl) > fabsf(dphi_hv_scaled));
    bool close_to_h_scaled = fabsf(dphi_hv_scaled) < HIP_PIO4_F && fabsf(dphi_hv_scaled) < fabsf(dphi_lv_scaled);
    bool close_to_l_scaled = fabsf(dphi_lv_scaled) < HIP_PIO4_F && fabsf(dphi_lv_scaled) < fabsf(dphi_hv_scaled);
    bool v_pos_pass_scaled = inside_hl_scaled || close_to_h_scaled || close_to_l_scaled;
    float omega = -999;
    if(!inside_hl_scaled && close_to_l_scaled && dphi_hv_scaled * dphi_hl < 0)
    {
        if(dphi_hl < 0)
            omega = (dphi_hv_scaled - 2 * HIP_PI_F) / dphi_hl;
        else if (dphi_hv_scaled < 0)
            omega = (dphi_hv_scaled + HIP_PI_F * 2.0f) / dphi_hl;
    }
    else
    {
        omega = dphi_hv_scaled / dphi_hl;
    }

    //p4 of vl vh
    float vh_scaled_pt =0;
    float vh_scaled_eta=0;
    float vh_scaled_phi=0;
    float vh_scaled_m  =0;

    float vl_scaled_pt =0;
    float vl_scaled_eta=0;
    float vl_scaled_phi=0;
    float vl_scaled_m  =0;
    float  m_tautau_scaled = -999;
    if(v_pos_pass_scaled)
    {
        
        if (!inside_hl_scaled && close_to_h_scaled) 
        {
            vh_scaled_pt  = met_scaled_pt * cosf(fabsf(dphi_hv_scaled));
            vh_scaled_eta = tau0_eta;
            vh_scaled_phi = tau0_phi; 
            vh_scaled_m   = 0;
        }
        else if (!inside_hl_scaled && close_to_l_scaled)
        {
            vl_scaled_pt = met_scaled_pt * cosf(fabsf(dphi_lv_scaled));
            vl_scaled_eta = lep0_eta;
            vl_scaled_phi = lep0_phi;
            vl_scaled_m   =  0;
        }
        else if (inside_hl_scaled)
        {
            vh_scaled_pt    = met_scaled_pt * cosf(fabsf(dphi_hv_scaled)) - met_scaled_pt * sinf(fabsf(dphi_hv_scaled)) * (1/tanf(fabsf(dphi_hl)));
            vh_scaled_eta   = tau0_eta;
            vh_scaled_phi   = tau0_phi; 
            vh_scaled_m     = 0;
            vl_scaled_pt    = met_scaled_pt * sinf(fabsf(dphi_hv_scaled)) / sinf(fabsf(dphi_hl));
            vl_scaled_eta   = lep0_eta;
            vl_scaled_phi   = lep0_phi;
            vl_scaled_m     = 0;
        }
        float  tautau_px = (getPx(vh_scaled_pt, vh_scaled_phi)+getPx(vl_scaled_pt, vl_scaled_phi)+getPx(tau0_pt, tau0_phi)+getPx(lep0_pt, lep0_phi));
        float  tautau_py = (getPy(vh_scaled_pt, vh_scaled_phi)+getPy(vl_scaled_pt, vl_scaled_phi)+getPy(tau0_pt, tau0_phi)+getPy(lep0_pt, lep0_phi));
        float  tautau_pz = (getPz(vh_scaled_pt, vh_scaled_eta)+getPz(vl_scaled_pt, vl_scaled_eta)+getPz(tau0_pt, tau0_eta)+getPz(lep0_pt, lep0_eta));
        float  tautau_e  = (getE(vh_scaled_pt, vh_scaled_eta, vh_scaled_m)+getE(vl_scaled_pt, vl_scaled_eta, vl_scaled_m)+getE(tau0_pt, tau0_eta, tau0_m)+getE(lep0_pt, lep0_eta, lep0_m));
        m_tautau_scaled = getMass(tautau_e, tautau_px, tautau_py, tautau_pz);
    }
    if(m_tautau_scaled > 0)
    {
        pass[ind] = 1;
        float score_m_bb = getPsiMbb(m_bb_scaled, Z_MASS, M_BB_SC);
        float score_m_tt = getPsiMtt(m_tautau_scaled, Z_MASS, M_TT_SC);
        float score_met  = getPsiMET(vl_scaled_pt, vh_scaled_pt, tau0_pt, lep0_pt, omega, MET_SC_1, MET_SC_2, MET_SC_3);
        float score_chi  = getPsiChi(chi0, chi1, CHI_SC);
        score[sizeof_array*1 + ind]  = score_m_bb;
        score[sizeof_array*2 + ind]  = score_m_tt;
        score[sizeof_array*3 + ind]  = score_met;
        score[sizeof_array*4 + ind]  = score_chi;
        score[ind] = sqrtf(score_chi * score_chi + score_m_bb * score_m_bb + score_m_tt * score_m_tt + score_met * score_met);
    }
}

std::vector<double> GPUScaleB(float bjet0_pt, float bjet0_eta,    float bjet0_phi,    float bjet0_m,
    float bjet1_pt, float bjet1_eta,    float bjet1_phi,    float bjet1_m,
    float lep0_pt,  float lep0_eta,     float lep0_phi,     float lep0_m,
    float tau0_pt,  float tau0_eta,     float tau0_phi,     float tau0_m,
    float met_pt,   float met_eta,      float met_phi,      float met_m){
    size_t N = 151;
    size_t NN = N*N;
    size_t size_bool  = NN * sizeof(bool);
    size_t size_float = 5 * NN * sizeof(float);

    //allocate arrays on GPU
    bool *pass_dev;
    hipMalloc((void**)&pass_dev, size_bool);

    float *score_dev;
    hipMalloc((void**)&score_dev, size_float);

    ////copy stuff to GPU
    // hipMemcpy(pass_dev,pass,size_bool,hipMemcpyHostToDevice);
    // hipMemcpy(score_dev,score,size_float,hipMemcpyHostToDevice);
    // hipMemcpy(score1_dev,score1,size_float,hipMemcpyHostToDevice);
    // hipMemcpy(score2_dev,score2,size_float,hipMemcpyHostToDevice);
    // hipMemcpy(score3_dev,score3,size_float,hipMemcpyHostToDevice);
    // hipMemcpy(score4_dev,score4,size_float,hipMemcpyHostToDevice);
    ////memset version
    // hipMemset(score_dev, 0, size_float);
    // hipMemset(score1_dev, 0, size_float);
    // hipMemset(score2_dev, 0, size_float);
    // hipMemset(score3_dev, 0, size_float);
    // hipMemset(score4_dev, 0, size_float);
    // hipMemset(pass_dev, 0, size_float);

    dim3 grid(N, N);
    mykernel<<<grid, 1>>>(  bjet0_pt,  bjet0_eta,     bjet0_phi,     bjet0_m,
                            bjet1_pt,  bjet1_eta,     bjet1_phi,     bjet1_m,
                            lep0_pt,   lep0_eta,      lep0_phi,      lep0_m,
                            tau0_pt,   tau0_eta,      tau0_phi,      tau0_m,
                            met_pt,    met_eta,       met_phi,       met_m,
                            pass_dev, score_dev, N);
    //hipDeviceSynchronize();
    
    std::vector<double> ret;
    double sf1 = -999;
    double sf2 = -999;
    double min_score = 200;
    double s1 = 100;
    double s2 = 100;
    double s3 = 100;
    double s4 = 100;

    // //CPU find min
    // bool pass[NN];
    // float score[NN*5];
    // hipMemcpy(pass,pass_dev,size_bool,hipMemcpyDeviceToHost);
    // hipMemcpy(score,score_dev,size_float,hipMemcpyDeviceToHost);
    // for(size_t i =0; i < NN; i++){
    //     if (pass[i])
    //     {
    //         if(score[i] < min_score)
    //         {
    //             min_score = score[i];

    //             s1 = score[NN + i];
    //             s2 = score[NN *2 + i];
    //             s3 = score[NN *3 + i];
    //             s4 = score[NN *4 + i];

    //             sf1 = 0.01 * (double)(i/151) + 0.5;
    //             sf2 = 0.01 * (double)(i%151) + 0.5;
    //         }
    //     }
    // }

    //GPU find min
    thrust::device_ptr<float> score_vec =  thrust::device_pointer_cast(score_dev);
    int min_offset = thrust::min_element(score_vec, score_vec + NN) - score_vec;

    min_score = *(score_vec + min_offset);
    if (min_score < 200)
    {
        s1 = *(score_vec + NN   + min_offset);
        s2 = *(score_vec + NN*2 + min_offset);
        s3 = *(score_vec + NN*3 + min_offset);
        s4 = *(score_vec + NN*4 + min_offset);
        sf1 = 0.01 * (double)(min_offset/N) + 0.5;
        sf2 = 0.01 * (double)(min_offset%N) + 0.5;
    }
    if (sf1 > 0 && sf2 > 0)
    {
        ret = {sf1, sf2, min_score, s1, s2, s3, s4};
    }
    
    hipFree(pass_dev);
    hipFree(score_dev);
    return ret;
}
