#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_math_constants.h>
#include "GPUScaleB.cuh"
#define Z_MASS 91.1876

__device__ float getPx(float pt, float phi){
    return pt * cosf(phi);
}

__device__ float getPy(float pt, float phi){
    return pt * sinf(phi);
}

__device__ float getPz(float pt, float eta){
    return pt * sinhf(eta);
}

__device__ float getPScaler(float pt, float eta){
    return pt * coshf(eta);
}

__device__ float getE(float pt, float eta, float m){
    float p = getPScaler(pt, eta);
    return sqrtf(m * m + p * p);
}

__device__ float getMass( float energy, float px, float py, float pz){

    return sqrtf(energy * energy - px * px - py * py - pz * pz);
}

__device__ float getdphi(float v1phi,  float v2phi)
{
    float ret;
    if(fabsf(v1phi - v2phi) > HIP_PI_F)
        ret = ((float)(v1phi > 0) -  (float)(v1phi <= 0)) * (2 * HIP_PI_F - fabsf(v1phi - v2phi));
    else
        ret = v2phi - v1phi;
    return ret;
}

__device__ float getPhiFromPxPy(float px, float py)
{
    return  (float)(px>0) * atanf(py / px) + 
            (float)(px<=0 && py > 0) * (atanf(py / px) + HIP_PI_F) + 
            (float)(px<=0 && py <= 0) * (atanf(py / px) - HIP_PI_F);
}

__global__ void mykernel(   float bjet0_pt, float bjet0_eta,    float bjet0_phi,    float bjet0_m,
                            float bjet1_pt, float bjet1_eta,    float bjet1_phi,    float bjet1_m,
                            float lep0_pt,  float lep0_eta,     float lep0_phi,     float lep0_m,
                            float tau0_pt,  float tau0_eta,     float tau0_phi,     float tau0_m,
                            float met_pt,   float met_eta,      float met_phi,      float met_m,
                            bool * pass, float* m_tt, int N)
{
    int i = (threadIdx.x + blockIdx.x * blockDim.x);
    
    int j = (threadIdx.y + blockIdx.y * blockDim.y);
    float chi0 = 0.5 + 0.01 * i;
    
    float chi1 = 0.5 + 0.01 * j;

    float bjet0_scaled_pt = bjet0_pt * chi0;
    float bjet1_scaled_pt = bjet1_pt * chi1;

    float met_scaled_px = getPx(met_pt, met_phi) - (getPx(bjet0_scaled_pt, bjet0_phi) - getPx(bjet0_pt, bjet0_phi)) - (getPx(bjet1_scaled_pt, bjet1_phi) - getPx(bjet1_pt, bjet1_phi));
    float met_scaled_py = getPy(met_pt, met_phi) - (getPy(bjet0_scaled_pt, bjet0_phi) - getPy(bjet0_pt, bjet0_phi)) - (getPy(bjet1_scaled_pt, bjet1_phi) - getPy(bjet1_pt, bjet1_phi));
    
    float bjet0_scaled_E = getE(bjet0_scaled_pt, bjet0_eta, bjet0_m);
    float bjet0_scaled_px = getPx(bjet0_scaled_pt, bjet0_phi);
    float bjet0_scaled_py = getPy(bjet0_scaled_pt, bjet0_phi);
    float bjet0_scaled_pz = getPz(bjet0_scaled_pt, bjet0_eta);

    float bjet1_scaled_E =  getE(bjet1_scaled_pt, bjet1_eta, bjet1_m);
    float bjet1_scaled_px = getPx(bjet1_scaled_pt, bjet1_phi);
    float bjet1_scaled_py = getPy(bjet1_scaled_pt, bjet1_phi);
    float bjet1_scaled_pz = getPz(bjet1_scaled_pt, bjet1_eta);


    float m_bb_scaled = getMass(bjet0_scaled_E + bjet1_scaled_E, 
                                bjet0_scaled_px + bjet1_scaled_px,
                                bjet0_scaled_py + bjet1_scaled_py,
                                bjet0_scaled_pz + bjet1_scaled_pz);   
    
    //printf("chi0:%f\tchi1:%f\tmbb:%f\n",chi0, chi1, m_bb_scaled);
    if(fabsf(m_bb_scaled - Z_MASS) < 1.0)
    {
        //printf("i:%d\tj:%d\tchi0:%f\tchi1:%f\tPASS1\n",i,j,chi0,chi1);
        float dphi_hl = getdphi(tau0_phi, lep0_phi);
        float met_scaled_phi = getPhiFromPxPy(met_scaled_px, met_scaled_py);
        float met_scaled_pt  = sqrtf(met_scaled_py * met_scaled_py + met_scaled_px * met_scaled_px);
        float dphi_hv_scaled = getdphi(tau0_phi, met_scaled_phi);
        float dphi_lv_scaled = getdphi(lep0_phi, met_scaled_phi);
        bool inside_hl_scaled = (dphi_hl * dphi_hv_scaled > 0) && (fabsf(dphi_hl) > fabsf(dphi_hv_scaled));
        bool close_to_h_scaled = fabsf(dphi_hv_scaled) < 0.17453292f;// 10 degree
        bool close_to_l_scaled = fabsf(dphi_lv_scaled) < 0.17453292f;
        bool v_pos_pass_scaled = inside_hl_scaled || close_to_h_scaled || close_to_l_scaled;
        if(v_pos_pass_scaled)
        {
            //printf("i:%d\tj:%d\tchi0:%f\tchi1:%f\tPASS2\n",i,j,chi0,chi1);
            float vh_scaled_pt =0;
            float vh_scaled_eta=0;
            float vh_scaled_phi=0;
            float vh_scaled_m  =0;

            float vl_scaled_pt =0;
            float vl_scaled_eta=0;
            float vl_scaled_phi=0;
            float vl_scaled_m  =0;
            if (!inside_hl_scaled && close_to_h_scaled) 
            {
                vh_scaled_pt  = met_scaled_pt * cosf(fabsf(dphi_hv_scaled));
                vh_scaled_eta = tau0_eta;
                vh_scaled_phi = tau0_phi; 
                vh_scaled_m   = 0;
            }
            else if (!inside_hl_scaled && close_to_l_scaled)
            {
                vl_scaled_pt = met_scaled_pt * cosf(fabsf(dphi_lv_scaled));
                vl_scaled_eta = lep0_eta;
                vl_scaled_phi = lep0_phi;
                vl_scaled_m   =  0;
            }
            else if (inside_hl_scaled)
            {
                vh_scaled_pt    = met_scaled_pt * cosf(fabsf(dphi_hv_scaled)) - met_scaled_pt * sinf(fabsf(dphi_hv_scaled)) * (1/tanf(fabsf(dphi_hl)));
                vh_scaled_eta   = tau0_eta;
                vh_scaled_phi   = tau0_phi; 
                vh_scaled_m     = 0;
                vl_scaled_pt    = met_scaled_pt * sinf(fabsf(dphi_hv_scaled)) / sinf(fabsf(dphi_hl));
                vl_scaled_eta   = lep0_eta;
                vl_scaled_phi   = lep0_phi;
                vl_scaled_m     = 0;
            }
            float  tautau_px = (getPx(vh_scaled_pt, vh_scaled_phi)+getPx(vl_scaled_pt, vl_scaled_phi)+getPx(tau0_pt, tau0_phi)+getPx(lep0_pt, lep0_phi));
            float  tautau_py = (getPy(vh_scaled_pt, vh_scaled_phi)+getPy(vl_scaled_pt, vl_scaled_phi)+getPy(tau0_pt, tau0_phi)+getPy(lep0_pt, lep0_phi));
            float  tautau_pz = (getPz(vh_scaled_pt, vh_scaled_eta)+getPz(vl_scaled_pt, vl_scaled_eta)+getPz(tau0_pt, tau0_eta)+getPz(lep0_pt, lep0_eta));
            float  tautau_e  = (getE(vh_scaled_pt, vh_scaled_eta, vh_scaled_m)+getE(vl_scaled_pt, vl_scaled_eta, vl_scaled_m)+getE(tau0_pt, tau0_eta, tau0_m)+getE(lep0_pt, lep0_eta, lep0_m));
            float  m_tautau_scaled = getMass(tautau_e, tautau_px, tautau_py, tautau_pz);
            float m_tautau_diff_scaled = fabsf(m_tautau_scaled - Z_MASS);

            if (m_tautau_diff_scaled < 3.0)
            {
                //printf("i:%d\tj:%d\tchi0:%f\tchi1:%f\tPASS3\n",i,j,chi0,chi1);
                pass[i*N + j] = 1;
                m_tt[i*N + j] = m_tautau_diff_scaled;
            }
        }
    }
}

std::vector<double> GPUScaleB(float bjet0_pt, float bjet0_eta,    float bjet0_phi,    float bjet0_m,
    float bjet1_pt, float bjet1_eta,    float bjet1_phi,    float bjet1_m,
    float lep0_pt,  float lep0_eta,     float lep0_phi,     float lep0_m,
    float tau0_pt,  float tau0_eta,     float tau0_phi,     float tau0_m,
    float met_pt,   float met_eta,      float met_phi,      float met_m){
    int N = 151;
    size_t size_bool  = N * N * sizeof(bool);
    size_t size_float = N * N * sizeof(float);
    bool *pass;
    pass = (bool*)malloc(size_bool);
    float *m_tt;
    m_tt = (float*)malloc(size_float);

    bool *pass_dev;
    hipMalloc((void**)&pass_dev, size_bool);

    float *m_tt_dev;
    hipMalloc((void**)&m_tt_dev, size_float);

    for(int i =0; i < N*N; i++)
    {
        pass[i]=0;
        m_tt[i]=0;
    }
    hipMemcpy(pass_dev,pass,size_bool,hipMemcpyHostToDevice);
    hipMemcpy(m_tt_dev,m_tt,size_float,hipMemcpyHostToDevice);

    dim3 grid(N, N);
    mykernel<<<grid, 1>>>(  bjet0_pt,  bjet0_eta,     bjet0_phi,     bjet0_m,
                            bjet1_pt,  bjet1_eta,     bjet1_phi,     bjet1_m,
                            lep0_pt,   lep0_eta,      lep0_phi,      lep0_m,
                            tau0_pt,   tau0_eta,      tau0_phi,      tau0_m,
                            met_pt,    met_eta,       met_phi,       met_m,
                            pass_dev, m_tt_dev, N);
    //hipDeviceSynchronize();
    hipMemcpy(pass,pass_dev,size_bool,hipMemcpyDeviceToHost);
    hipMemcpy(m_tt,m_tt_dev,size_float,hipMemcpyDeviceToHost);
    hipFree(pass_dev);
    hipFree(m_tt_dev);
    double min_mtt_diff = 100000;
    std::vector<double> ret;
    double sf1 = -999;
    double sf2 = -999;
    for(int i =0; i < N * N; i++){
        if (pass[i])
        {
            if(m_tt[i] < min_mtt_diff)
            {
                min_mtt_diff = m_tt[i];
                sf1 = 0.01 * (double)(i/151) + 0.5;
                sf2 = 0.01 * (double)(i%151) + 0.5;
            }
        }
    }
    if (sf1 > 0 && sf2 > 0)
    {
        ret.push_back(sf1);
        ret.push_back(sf2);
    }
    free(pass);
    free(m_tt);

    return ret;
}
