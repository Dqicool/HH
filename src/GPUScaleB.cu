#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_math_constants.h>
#include "GPUScaleB.cuh"
#define Z_MASS 91.1876

#define M_BB_SC     62.5
#define M_TT_SC     62.5
#define MET_SC_1    0.008
#define MET_SC_2    0.0005
#define MET_SC_3    0.000125
#define CHI_SC      0.04

__device__ float getPx(float pt, float phi){
    return pt * cosf(phi);
}

__device__ float getPy(float pt, float phi){
    return pt * sinf(phi);
}

__device__ float getPz(float pt, float eta){
    return pt * sinhf(eta);
}

__device__ float getPScaler(float pt, float eta){
    return pt * coshf(eta);
}

__device__ float getE(float pt, float eta, float m){
    float p = getPScaler(pt, eta);
    return sqrtf(m * m + p * p);
}

__device__ float getMass( float energy, float px, float py, float pz){

    return sqrtf(energy * energy - px * px - py * py - pz * pz);
}

__device__ float getdphi(float v1phi,  float v2phi)
{
    float ret;
    if(fabsf(v1phi - v2phi) > HIP_PI_F)
        ret = ((float)(v1phi > 0) -  (float)(v1phi <= 0)) * (2 * HIP_PI_F - fabsf(v1phi - v2phi));
    else
        ret = v2phi - v1phi;
    return ret;
}

__device__ float getPhiFromPxPy(float px, float py)
{
    return  (float)(px>0) * atanf(py / px) + 
            (float)(px<=0 && py > 0) * (atanf(py / px) + HIP_PI_F) + 
            (float)(px<=0 && py <= 0) * (atanf(py / px) - HIP_PI_F);
}

__device__ float getPsiMbb(float m_bb, float m_baseline, float scale_const_bb)
{
    return (m_bb - m_baseline) * (m_bb - m_baseline) / scale_const_bb;
}

__device__ float getPsiMtt(float m_tt, float m_baseline, float scale_const_tt)
{
    return (m_tt - m_baseline) * (m_tt - m_baseline) / scale_const_tt;
}

__device__ float getPsiMET(float vl_pt, float vh_pt, float tau_pt, float lep_pt, float omega,float sc_1, float sc_2, float sc_3)
{
    float f_tau_l = vl_pt / (vl_pt + lep_pt);
    float f_tau_h = vh_pt / (vh_pt + tau_pt);
    float A_1 = 0.5 * ((f_tau_l - f_tau_h) / (f_tau_l + f_tau_h) + 1);
    float ret = ((float)(A_1 > 0 && A_1 <= 0.8) * (A_1 - 0.8) * (A_1 - 0.8) / sc_1) +
                ((float)(A_1 > 0.8 && A_1 <= 1) * 0) +
                ((float)(omega > 1)               * (omega - 1) * (omega - 1) / sc_2) +
                ((float)(omega <= 0)              * (80 + (omega * omega / sc_3)));
    return ret;
}

__device__ float getPsiChi(float chi0, float chi1, float sc_chi)
{
    float log2_chi0 = log2f(chi0);
    float log2_chi1 = log2f(chi1);
    float r2 = log2_chi0 * log2_chi0 + log2_chi1 * log2_chi1;
    return r2 / sc_chi;
}

__global__ void mykernel(   float bjet0_pt, float bjet0_eta,    float bjet0_phi,    float bjet0_m,
                            float bjet1_pt, float bjet1_eta,    float bjet1_phi,    float bjet1_m,
                            float lep0_pt,  float lep0_eta,     float lep0_phi,     float lep0_m,
                            float tau0_pt,  float tau0_eta,     float tau0_phi,     float tau0_m,
                            float met_pt,   float met_eta,      float met_phi,      float met_m,
                            bool * pass, float* score, float* score1, float* score2, float* score3, float* score4, int N)
{
    int i = (threadIdx.x + blockIdx.x * blockDim.x);
    int j = (threadIdx.y + blockIdx.y * blockDim.y);
    float chi0 = 0.5 + 0.01 * i;
    float chi1 = 0.5 + 0.01 * j;
    // initialise
    pass[i*N + j] = 0;
    score1[i*N + j]  = -999;
    score2[i*N + j]  = -999;
    score3[i*N + j]  = -999;
    score4[i*N + j]  = -999;
    score[i*N + j]   = -999;
    //mbb calculation
    float bjet0_scaled_pt = bjet0_pt * chi0;
    float bjet1_scaled_pt = bjet1_pt * chi1;

    float met_scaled_px = getPx(met_pt, met_phi) - (getPx(bjet0_scaled_pt, bjet0_phi) - getPx(bjet0_pt, bjet0_phi)) - (getPx(bjet1_scaled_pt, bjet1_phi) - getPx(bjet1_pt, bjet1_phi));
    float met_scaled_py = getPy(met_pt, met_phi) - (getPy(bjet0_scaled_pt, bjet0_phi) - getPy(bjet0_pt, bjet0_phi)) - (getPy(bjet1_scaled_pt, bjet1_phi) - getPy(bjet1_pt, bjet1_phi));
    
    float bjet0_scaled_E = getE(bjet0_scaled_pt, bjet0_eta, bjet0_m);
    float bjet0_scaled_px = getPx(bjet0_scaled_pt, bjet0_phi);
    float bjet0_scaled_py = getPy(bjet0_scaled_pt, bjet0_phi);
    float bjet0_scaled_pz = getPz(bjet0_scaled_pt, bjet0_eta);

    float bjet1_scaled_E =  getE(bjet1_scaled_pt, bjet1_eta, bjet1_m);
    float bjet1_scaled_px = getPx(bjet1_scaled_pt, bjet1_phi);
    float bjet1_scaled_py = getPy(bjet1_scaled_pt, bjet1_phi);
    float bjet1_scaled_pz = getPz(bjet1_scaled_pt, bjet1_eta);


    float m_bb_scaled = getMass(bjet0_scaled_E + bjet1_scaled_E, 
                                bjet0_scaled_px + bjet1_scaled_px,
                                bjet0_scaled_py + bjet1_scaled_py,
                                bjet0_scaled_pz + bjet1_scaled_pz);   
    
    //omega and other
    float dphi_hl = getdphi(tau0_phi, lep0_phi);
    float met_scaled_phi = getPhiFromPxPy(met_scaled_px, met_scaled_py);
    float met_scaled_pt  = sqrtf(met_scaled_py * met_scaled_py + met_scaled_px * met_scaled_px);
    float dphi_hv_scaled = getdphi(tau0_phi, met_scaled_phi);
    float dphi_lv_scaled = getdphi(lep0_phi, met_scaled_phi);
    bool inside_hl_scaled = (dphi_hl * dphi_hv_scaled > 0) && (fabsf(dphi_hl) > fabsf(dphi_hv_scaled));
    bool close_to_h_scaled = fabsf(dphi_hv_scaled) < HIP_PIO4_F && fabsf(dphi_hv_scaled) < fabsf(dphi_lv_scaled);
    bool close_to_l_scaled = fabsf(dphi_lv_scaled) < HIP_PIO4_F && fabsf(dphi_lv_scaled) < fabsf(dphi_hv_scaled);
    bool v_pos_pass_scaled = inside_hl_scaled || close_to_h_scaled || close_to_l_scaled;
    float omega = -999;
    if(!inside_hl_scaled && close_to_l_scaled && dphi_hv_scaled * dphi_hl < 0)
    {
        if(dphi_hl < 0)
            omega = (dphi_hv_scaled - 2 * HIP_PI_F) / dphi_hl;
        else if (dphi_hv_scaled < 0)
            omega = (dphi_hv_scaled + HIP_PI_F * 2.0f) / dphi_hl;
    }
    else
    {
        omega = dphi_hv_scaled / dphi_hl;
    }

    //p4 of vl vh
    float vh_scaled_pt =0;
    float vh_scaled_eta=0;
    float vh_scaled_phi=0;
    float vh_scaled_m  =0;

    float vl_scaled_pt =0;
    float vl_scaled_eta=0;
    float vl_scaled_phi=0;
    float vl_scaled_m  =0;
    float  m_tautau_scaled = -999;
    if(v_pos_pass_scaled)
    {
        
        if (!inside_hl_scaled && close_to_h_scaled) 
        {
            vh_scaled_pt  = met_scaled_pt * cosf(fabsf(dphi_hv_scaled));
            vh_scaled_eta = tau0_eta;
            vh_scaled_phi = tau0_phi; 
            vh_scaled_m   = 0;
        }
        else if (!inside_hl_scaled && close_to_l_scaled)
        {
            vl_scaled_pt = met_scaled_pt * cosf(fabsf(dphi_lv_scaled));
            vl_scaled_eta = lep0_eta;
            vl_scaled_phi = lep0_phi;
            vl_scaled_m   =  0;
        }
        else if (inside_hl_scaled)
        {
            vh_scaled_pt    = met_scaled_pt * cosf(fabsf(dphi_hv_scaled)) - met_scaled_pt * sinf(fabsf(dphi_hv_scaled)) * (1/tanf(fabsf(dphi_hl)));
            vh_scaled_eta   = tau0_eta;
            vh_scaled_phi   = tau0_phi; 
            vh_scaled_m     = 0;
            vl_scaled_pt    = met_scaled_pt * sinf(fabsf(dphi_hv_scaled)) / sinf(fabsf(dphi_hl));
            vl_scaled_eta   = lep0_eta;
            vl_scaled_phi   = lep0_phi;
            vl_scaled_m     = 0;
        }
        float  tautau_px = (getPx(vh_scaled_pt, vh_scaled_phi)+getPx(vl_scaled_pt, vl_scaled_phi)+getPx(tau0_pt, tau0_phi)+getPx(lep0_pt, lep0_phi));
        float  tautau_py = (getPy(vh_scaled_pt, vh_scaled_phi)+getPy(vl_scaled_pt, vl_scaled_phi)+getPy(tau0_pt, tau0_phi)+getPy(lep0_pt, lep0_phi));
        float  tautau_pz = (getPz(vh_scaled_pt, vh_scaled_eta)+getPz(vl_scaled_pt, vl_scaled_eta)+getPz(tau0_pt, tau0_eta)+getPz(lep0_pt, lep0_eta));
        float  tautau_e  = (getE(vh_scaled_pt, vh_scaled_eta, vh_scaled_m)+getE(vl_scaled_pt, vl_scaled_eta, vl_scaled_m)+getE(tau0_pt, tau0_eta, tau0_m)+getE(lep0_pt, lep0_eta, lep0_m));
        m_tautau_scaled = getMass(tautau_e, tautau_px, tautau_py, tautau_pz);
    }
    if(m_tautau_scaled > 0)
    {
        pass[i*N + j] = 1;
        float score_m_bb = getPsiMbb(m_bb_scaled, Z_MASS, M_BB_SC);
        float score_m_tt = getPsiMtt(m_tautau_scaled, Z_MASS, M_TT_SC);
        float score_met  = getPsiMET(vl_scaled_pt, vh_scaled_pt, tau0_pt, lep0_pt, omega, MET_SC_1, MET_SC_2, MET_SC_3);
        float score_chi  = getPsiChi(chi0, chi1, CHI_SC);
        score1[i*N + j]  = score_m_bb;
        score2[i*N + j]  = score_m_tt;
        score3[i*N + j]  = score_met;
        score4[i*N + j]  = score_chi;
        score[i*N + j] = sqrtf(score_chi * score_chi + score_m_bb * score_m_bb + score_m_tt * score_m_tt + score_met * score_met);
    }
}

std::vector<double> GPUScaleB(float bjet0_pt, float bjet0_eta,    float bjet0_phi,    float bjet0_m,
    float bjet1_pt, float bjet1_eta,    float bjet1_phi,    float bjet1_m,
    float lep0_pt,  float lep0_eta,     float lep0_phi,     float lep0_m,
    float tau0_pt,  float tau0_eta,     float tau0_phi,     float tau0_m,
    float met_pt,   float met_eta,      float met_phi,      float met_m){
    int N = 151;
    size_t size_bool  = N * N * sizeof(bool);
    size_t size_float = N * N * sizeof(float);
    //allocate arrays on CPU
    bool pass[N*N];
    
    float score[N*N];
    float score1[N*N];
    float score2[N*N];
    float score3[N*N];
    float score4[N*N];
    //allocate arrays on GPU
    bool *pass_dev;
    hipMalloc((void**)&pass_dev, size_bool);

    float *score_dev, *score1_dev, *score2_dev, *score3_dev, *score4_dev; 
    hipMalloc((void**)&score_dev, size_float);
    hipMalloc((void**)&score1_dev, size_float);
    hipMalloc((void**)&score2_dev, size_float);
    hipMalloc((void**)&score3_dev, size_float);
    hipMalloc((void**)&score4_dev, size_float);
    //copy stuff to GPU
    // hipMemcpy(pass_dev,pass,size_bool,hipMemcpyHostToDevice);
    // hipMemcpy(score_dev,score,size_float,hipMemcpyHostToDevice);
    // hipMemcpy(score1_dev,score1,size_float,hipMemcpyHostToDevice);
    // hipMemcpy(score2_dev,score2,size_float,hipMemcpyHostToDevice);
    // hipMemcpy(score3_dev,score3,size_float,hipMemcpyHostToDevice);
    // hipMemcpy(score4_dev,score4,size_float,hipMemcpyHostToDevice);
    ////memset version
    // hipMemset(score_dev, 0, size_float);
    // hipMemset(score1_dev, 0, size_float);
    // hipMemset(score2_dev, 0, size_float);
    // hipMemset(score3_dev, 0, size_float);
    // hipMemset(score4_dev, 0, size_float);
    // hipMemset(pass_dev, 0, size_float);

    dim3 grid(N, N);
    mykernel<<<grid, 1>>>(  bjet0_pt,  bjet0_eta,     bjet0_phi,     bjet0_m,
                            bjet1_pt,  bjet1_eta,     bjet1_phi,     bjet1_m,
                            lep0_pt,   lep0_eta,      lep0_phi,      lep0_m,
                            tau0_pt,   tau0_eta,      tau0_phi,      tau0_m,
                            met_pt,    met_eta,       met_phi,       met_m,
                            pass_dev, score_dev, score1_dev, score2_dev, score3_dev, score4_dev, N);
    //hipDeviceSynchronize();
    hipMemcpy(pass,pass_dev,size_bool,hipMemcpyDeviceToHost);
    hipMemcpy(score,score_dev,size_float,hipMemcpyDeviceToHost);
    hipMemcpy(score1,score1_dev,size_float,hipMemcpyDeviceToHost);
    hipMemcpy(score2,score2_dev,size_float,hipMemcpyDeviceToHost);
    hipMemcpy(score3,score3_dev,size_float,hipMemcpyDeviceToHost);
    hipMemcpy(score4,score4_dev,size_float,hipMemcpyDeviceToHost);
    hipFree(pass_dev);
    hipFree(score_dev);
    hipFree(score1_dev);
    hipFree(score2_dev);
    hipFree(score3_dev);
    hipFree(score4_dev);

    std::vector<double> ret;
    double sf1 = -999;
    double sf2 = -999;
    double min_score = 200;
    double s1 = 100;
    double s2 = 100;
    double s3 = 100;
    double s4 = 100;
    for(int i =0; i < N * N; i++){
        if (pass[i])
        {
            if(score[i] < min_score)
            {
                min_score = score[i];

                s1 = score1[i];
                s2 = score2[i];
                s3 = score3[i];
                s4 = score4[i];

                sf1 = 0.01 * (double)(i/151) + 0.5;
                sf2 = 0.01 * (double)(i%151) + 0.5;
            }
        }
    }
    if (sf1 > 0 && sf2 > 0)
    {
        ret.push_back(sf1);
        ret.push_back(sf2);
        ret.push_back(min_score);
        ret.push_back(s1);
        ret.push_back(s2);
        ret.push_back(s3);
        ret.push_back(s4);
    }

    return ret;
}
